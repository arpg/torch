#include "hip/hip_runtime.h"
#include <optix.h>
#include <torch/device/Random.h>

typedef rtCallableProgramX<uint2(const float2&, float&)> SampleFunction;

rtDeclareVariable(unsigned int, launchIndex, rtLaunchIndex, );
rtDeclareVariable(SampleFunction, Sample2D, , );
rtBuffer<unsigned int> offsets;
rtBuffer<unsigned int> counts;
rtBuffer<float> pdfs;

TORCH_DEVICE unsigned int GetBufferIndex(const uint2& index)
{
  return offsets[index.x] + index.y;
}

RT_PROGRAM void Sample()
{
  float pdf;
  unsigned int seed = torch::init_seed<16>(launchIndex, 0);
  const uint2 index = Sample2D(torch::randf2(seed), pdf);
  const unsigned int bufferIndex = GetBufferIndex(index);
  atomicAdd(&counts[bufferIndex], 1);
  pdfs[bufferIndex] = pdf;
}