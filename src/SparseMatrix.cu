#include "hip/hip_runtime.h"
#include <torch/device/Core.h>

rtBuffer<float3, 1> values;
rtBuffer<unsigned int, 1> rowOffsets;
rtBuffer<unsigned int, 1> colIndices;

RT_CALLABLE_PROGRAM void Add(uint row, uint col, float3 value)
{
  unsigned int begin = rowOffsets[row];
  unsigned int end = rowOffsets[row + 1];

  if (begin < end)
  {
    unsigned int index = begin;

    while (begin < end)
    {
      index = (begin + end) / 2;

      if (col == colIndices[index])
      {
        values[index] += value;
        return;
      }

      (col < colIndices[index]) ? end = index : begin = index + 1;
    }
  }
}