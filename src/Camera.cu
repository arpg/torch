#include "hip/hip_runtime.h"
#include <optix.h>
#include <torch/device/Core.h>
#include <torch/device/Camera.h>
#include <torch/device/Random.h>
#include <torch/device/Ray.h>

rtDeclareVariable(uint2, pixelIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, imageSize, rtLaunchDim, );

rtDeclareVariable(rtObject, sceneRoot, , );
rtDeclareVariable(float, sceneEpsilon, , );

rtDeclareVariable(unsigned int, sampleCount, , );
rtDeclareVariable(torch::CameraData, camera, , );
rtBuffer<float3, 2> buffer;

static __inline__ __device__
void GetDirection(float3& direction, unsigned int& seed)
{
  const float2 size = make_float2(imageSize);
  const float2 pixel = make_float2(pixelIndex) + torch::randf2(seed);
  const float2 ratio = (pixel / size) - camera.center;
  direction = ratio.x * camera.u + ratio.y * camera.v + camera.w;
  direction = normalize(direction);
}

TORCH_DEVICE void InitializeRay(optix::Ray& ray, torch::RadianceData& data)
{
  ray.origin = camera.position;
  ray.tmin = sceneEpsilon;
  ray.tmax = RT_DEFAULT_MAX;
  ray.ray_type = torch::RAY_TYPE_RADIANCE;
}

TORCH_DEVICE unsigned int InitializeSeed()
{
  unsigned int a = pixelIndex.x;
  unsigned int b = pixelIndex.y;
  return torch::init_seed<16>(a, b);
}

RT_PROGRAM void Capture()
{
  optix::Ray ray;
  torch::RadianceData data;
  unsigned int seed;

  seed = InitializeSeed();
  InitializeRay(ray, data);
  data.radiance = make_float3(0, 0, 0);

  for (unsigned int i = 0; i < sampleCount; ++i)
  {
    data.bounce.origin = make_float3(0, 0, 0);
    data.bounce.direction = make_float3(0, 0, 0);
    data.bounce.throughput = make_float3(0, 0, 0);
    data.depth = 0;

    GetDirection(ray.direction, seed);
    data.seed = seed;
    data.throughput = make_float3(1.0f / sampleCount);
    rtTrace(sceneRoot, ray, data);
    InitializeRay(ray, data);
    seed = data.seed;

    if (dot(data.bounce.direction, data.bounce.direction) > 0)
    {
      data.depth = 1;
      ray.origin = data.bounce.origin;
      ray.direction = data.bounce.direction;
      data.throughput = data.bounce.throughput;
      rtTrace(sceneRoot, ray, data);
      InitializeRay(ray, data);
      seed = data.seed;
    }
  }

  buffer[pixelIndex] = data.radiance;
}