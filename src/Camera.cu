#include "hip/hip_runtime.h"
#include <optix.h>
#include <torch/Ray.h>

rtDeclareVariable(uint2, pixelIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, imageSize, rtLaunchDim, );

rtDeclareVariable(rtObject, sceneRoot, , );
rtDeclareVariable(float, sceneEpsilon, , );

rtDeclareVariable(float3, position, , );
rtDeclareVariable(float2, center, , );
rtDeclareVariable(float3, u, , );
rtDeclareVariable(float3, v, , );
rtDeclareVariable(float3, w, , );
rtBuffer<float3, 2> buffer;

#define TORCH_DEVICE static __device__ __inline__

static __inline__ __device__
void GetDirection(float3& direction)
{
  const float2 pixel = make_float2(pixelIndex);
  const float2 size = make_float2(imageSize);
  const float2 ratio = (pixel / size) - center;
  direction = ratio.x * u + ratio.y * v + w;
  direction = normalize(direction);
}

TORCH_DEVICE void InitializeRay(optix::Ray& ray, torch::RadianceData& data)
{
  ray.origin = position;
  ray.tmin = sceneEpsilon;
  ray.tmax = RT_DEFAULT_MAX;
  ray.ray_type = torch::RAY_TYPE_RADIANCE;
  data.radiance = make_float3(0, 0, 0);
  GetDirection(ray.direction);
}

RT_PROGRAM void Capture()
{
  optix::Ray ray;
  torch::RadianceData data;
  InitializeRay(ray, data);
  rtTrace(sceneRoot, ray, data);
  buffer[pixelIndex] = data.radiance;
}