#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include <torch/device/Geometry.h>

rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtBuffer<float3, 1> vertices;
rtBuffer<float3, 1> normals;
rtBuffer<uint3, 1> faces;

// TORCH_DEVICE bool ReportIntersect(float t)
// {
//   // transform ray to local space
//   const float3 origin = PointToLocal(ray.origin);
//   const float3 direction = NormalToLocal(ray.direction);
//
//   // compute scaled hit point
//   const float3 localPoint = origin + t * direction;
//   const float3 worldPoint = PointToWorld(localPoint);
//
//   // compute time to scaled hit point
//   const float3 delta = worldPoint - ray.origin;
//   const int sign = (dot(delta, ray.direction) < 0) ? -1 : 1;
//   t = sign * sqrt(dot(delta, delta));
//
//   // check if valid intersect
//   if (rtPotentialIntersection(t))
//   {
//     // compute scaled surface normal
//     geometricNormal = NormalToWorld(localPoint);
//     shadingNormal = geometricNormal;
//     rtReportIntersection(0);
//     return true;
//   }
//
//   // invalid intersect
//   return false;
// }

RT_PROGRAM void Intersect(int index)
{
  const uint3& face = faces[index];
  const float3& v0 = vertices[face.x];
  const float3& v1 = vertices[face.y];
  const float3& v2 = vertices[face.z];

  optix::Ray triRay;
  triRay.origin = PointToLocal(ray.origin);
  triRay.direction = normalize(VectorToLocal(ray.direction));
  triRay.tmax = RT_DEFAULT_MAX;
  triRay.tmin = 0;

  float3 n;
  float t;
  float beta;
  float gamma;

  bool hit = intersect_triangle(triRay, v0, v1, v2, n, t, beta, gamma);

  if (hit)
  {
    // compute scaled hit point
    const float3 localPoint = triRay.origin + t * normalize(triRay.direction);
    const float3 worldPoint = PointToWorld(localPoint);

    // compute time to scaled hit point
    const float3 delta = worldPoint - ray.origin;
    const int sign = (dot(delta, ray.direction) < 0) ? -1 : 1;
    t = sign * sqrt(dot(delta, delta));

    // check if valid intersect
    if (rtPotentialIntersection(t))
    {
      // compute scaled surface normal
      geometricNormal = NormalToWorld(n);

      if (normals.size())
      {
        const float alpha = 1 - beta - gamma;
        shadingNormal = alpha * normals[face.x] + beta * normals[face.y] + gamma * normals[face.z];
      }
      else
      {
        shadingNormal = geometricNormal;
      }
      rtReportIntersection(0);
    }
  }
}

RT_PROGRAM void GetBounds(int index, float bounds[6])
{
  const uint3& face = faces[index];
  const float3& v0 = vertices[face.x];
  const float3& v1 = vertices[face.y];
  const float3& v2 = vertices[face.z];

  const float3 bmin = fminf(fminf(v0, v1), v2);
  const float3 bmax = fmaxf(fmaxf(v0, v1), v2);
  BoundsToWorld(bmin, bmax, bounds);
}