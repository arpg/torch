#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include <torch/device/Ray.h>
#include <torch/device/Transform.h>

rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(float2, triScales, attribute triScales, );
rtDeclareVariable(uint3, triFace, attribute triFace, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(torch::DepthData, depthData, rtPayload, );
rtBuffer<float3, 1> vertices;
rtBuffer<float3, 1> normals;
rtBuffer<uint3, 1> faces;

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );

RT_PROGRAM void Intersect(int index)
{
  const uint3& face = faces[index];
  const float3& v0 = vertices[face.x];
  const float3& v1 = vertices[face.y];
  const float3& v2 = vertices[face.z];

  optix::Ray triRay;
  triRay.origin = PointToLocal(ray.origin);
  triRay.direction = normalize(VectorToLocal(ray.direction));
  triRay.direction = normalize(VectorToLocal(ray.direction));
  triRay.tmax = RT_DEFAULT_MAX;
  triRay.tmin = 0.0f;

  float3 n;
  float t;
  float beta;
  float gamma;

  // bool hit = IntersectTriangle(triRay, v0, v1, v2, n, t, beta, gamma);
  bool hit = intersect_triangle(triRay, v0, v1, v2, n, t, beta, gamma);
  n = normalize(n);
  if (dot(n, -ray.direction) < 0.0f) n = -n;

  if (hit)
  {
    // compute scaled hit point
    const float3 localPoint = triRay.origin + t * normalize(triRay.direction);
    const float3 worldPoint = PointToWorld(localPoint);

    // compute time to scaled hit point
    const float3 delta = worldPoint - ray.origin;
    const int sign = (dot(delta, ray.direction) < 0) ? -1 : 1;
    t = sign * sqrtf(dot(delta, delta));

    // check if valid intersect
    if (rtPotentialIntersection(t))
    {
      // compute scaled surface normal
      geometricNormal = NormalToWorld(n);
      triScales = make_float2(beta, gamma);
      triFace = face;

      if (normals.size())
      {
        const float alpha = 1.0f - beta - gamma;
        shadingNormal = alpha * normals[face.x] + beta * normals[face.y] + gamma * normals[face.z];
        shadingNormal = faceforward(shadingNormal, -ray.direction, geometricNormal);
        shadingNormal = normalize(shadingNormal);

        if (isnan(shadingNormal.x) ||
            isnan(shadingNormal.y) ||
            isnan(shadingNormal.z))
        {
          shadingNormal = geometricNormal;
        }
      }
      else
      {
        shadingNormal = geometricNormal;
      }

      rtReportIntersection(0);
    }
  }
}

RT_PROGRAM void GetBounds(int index, float bounds[6])
{
  const uint3& face = faces[index];
  const float3& v0 = vertices[face.x];
  const float3& v1 = vertices[face.y];
  const float3& v2 = vertices[face.z];

  const float3 bmin = fminf(fminf(v0, v1), v2);
  const float3 bmax = fmaxf(fmaxf(v0, v1), v2);
  BoundsToWorld(bmin, bmax, bounds);
}
