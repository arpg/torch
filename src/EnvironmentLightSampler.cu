#include "hip/hip_runtime.h"
#include <torch/device/Light.h>
#include <torch/device/Random.h>
#include <torch/device/Visibility.h>

typedef rtCallableProgramX<uint(float, float&)> Distribution1D;
typedef rtCallableProgramId<uint2(const float2&, float&)> Distribution2D;

rtDeclareVariable(Distribution1D, GetLightIndex, , );
rtBuffer<Distribution2D> SampleLight;
rtBuffer<optix::Matrix3x3> rotations;
rtBuffer<uint> lightOffsets;
rtBuffer<uint> rowOffsets;
rtBuffer<float3> radiance;

TORCH_DEVICE uint GetRowCount(uint light)
{
  return lightOffsets[light + 1] - lightOffsets[light];
}

TORCH_DEVICE uint GetColumnCount(uint light, uint row)
{
  const uint lightOffset = lightOffsets[light];
  const uint rowOffset = rowOffsets[lightOffset] + row;
  return rowOffsets[rowOffset + 1] - rowOffsets[rowOffset];
}

TORCH_DEVICE float3 GetRadiance(uint light, uint row, uint col)
{
  const uint lightOffset = lightOffsets[light];
  const uint rowOffset = rowOffsets[lightOffset + row];
  return radiance[rowOffset + col];
}

TORCH_DEVICE void GetDirection(uint light, uint row, uint col, float3& dir)
{
  const uint rowCount = GetRowCount(light);
  const uint colCount = GetColumnCount(light, row);
  const float rowRadians = row * M_PIf / (rowCount - 1);
  const float colRadians = col * 2 * M_PIf / colCount;
  const float rowRadius = sinf(rowRadians);
  dir.x = rowRadius * sinf(colRadians);
  dir.z = rowRadius * cosf(colRadians);
  dir.y = cosf(rowRadians);
  dir = -normalize(dir);
}

RT_CALLABLE_PROGRAM void Sample(torch::LightSample& sample)
{
  const float rand = torch::randf(sample.seed);
  const uint light = GetLightIndex(rand, sample.pdf);

  float dirPdf;
  const float2 uv = torch::randf2(sample.seed);
  const uint2 index = SampleLight[light](uv, dirPdf);
  GetDirection(light, index.x, index.y, sample.direction);

  sample.direction = rotations[light] * sample.direction;
  sample.direction = normalize(sample.direction);
  sample.radiance = GetRadiance(light, index.x, index.y);

  sample.tmax = RT_DEFAULT_MAX;
  sample.pdf *= dirPdf;

  if (!torch::IsVisible(sample)) sample.radiance = make_float3(0, 0, 0);
}