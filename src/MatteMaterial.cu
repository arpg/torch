#include "hip/hip_runtime.h"
#include <optix.h>
#include <torch/device/Light.h>
#include <torch/device/Random.h>
#include <torch/device/Ray.h>
#include <torch/device/Sampling.h>

rtDeclareVariable(rtObject, sceneRoot, , );
rtDeclareVariable(float, sceneEpsilon, , );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(torch::RadianceData, rayData, rtPayload, );
rtDeclareVariable(float, hitDist, rtIntersectionDistance, );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(uint3, triFace, attribute triFace, );
rtDeclareVariable(float2, triScales, attribute triScales, );
rtBuffer<float3, 1> albedos;

rtDeclareVariable(optix::Ray, sray, rtCurrentRay, );
rtDeclareVariable(torch::ShadowData, srayData, rtPayload, );

typedef rtCallableProgramX<void(torch::LightSample&)> SampleLightFunction;
rtDeclareVariable(SampleLightFunction, SampleLights, , );

typedef rtCallableProgramId<void(uint, uint, float)> JacobianAddFunction;
rtBuffer<JacobianAddFunction, 1> AddToAlbedoJacobian;

TORCH_DEVICE float3 GetAlbedo()
{
  const uint x = (triFace.x >= albedos.size()) ? albedos.size() - 1 : triFace.x;
  const uint y = (triFace.y >= albedos.size()) ? albedos.size() - 1 : triFace.y;
  const uint z = (triFace.z >= albedos.size()) ? albedos.size() - 1 : triFace.z;
  const float b = fminf(1, triScales.x);
  const float c = fminf(1 - b, triScales.y);
  const float a = 1 - b - c;
  return a * albedos[x] + b * albedos[y] + c * albedos[z];
}

TORCH_DEVICE optix::Matrix3x3 NormalToRotation(const float3& n)
{
 float3 u = make_float3(1, 0, 0);
 float3 v = make_float3(0, 0, 1);
 if (dot(u, n) < dot(v, n)) u = v;
 v = normalize(cross(n, u));
 u = normalize(cross(n, v));

 optix::Matrix3x3 R;
 R.setCol(0, u);
 R.setCol(1, v);
 R.setCol(2, n);
 return R;
}

TORCH_DEVICE void SampleBrdf(torch::BrdfSample& sample)
{
  float3 direction;
  torch::SampleHemisphereCosine(torch::randf2(sample.seed), direction);
  sample.pdf = direction.z / M_PIf;
  sample.direction = NormalToRotation(sample.normal) * direction;
  sample.throughput = GetAlbedo() / M_PIf;
}

RT_PROGRAM void ClosestHit()
{
  torch::LightSample sample;
  sample.origin = ray.origin + hitDist * ray.direction;
  sample.tmin = sceneEpsilon;
  sample.seed = rayData.seed;
  sample.normal = geometricNormal;
  const unsigned int lightSamples = 16;

  for (unsigned int i = 0; i < lightSamples; ++i)
  {
    SampleLights(sample);
    rayData.seed = sample.seed;

    if (sample.radiance.x > 0 || sample.radiance.y > 0 || sample.radiance.z > 0)
    {
      const float3 brdf = GetAlbedo() / M_PIf;
      const float theta = dot(shadingNormal, sample.direction);
      rayData.radiance += (rayData.throughput * brdf * sample.radiance * theta / sample.pdf) / lightSamples;
    }
  }

  torch::BrdfSample brdfSample;
  brdfSample.normal = shadingNormal;
  brdfSample.seed = rayData.seed;
  SampleBrdf(brdfSample);

  float theta = dot(shadingNormal, brdfSample.direction);
  rayData.bounce.origin = sample.origin;
  rayData.bounce.direction = brdfSample.direction;
  rayData.bounce.throughput = theta * rayData.throughput * brdfSample.throughput / brdfSample.pdf;
}

RT_PROGRAM void AnyHit()
{
  srayData.occluded = true;
  rtTerminateRay();
}
