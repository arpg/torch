#include "hip/hip_runtime.h"
#include <optix.h>
#include <torch/device/Light.h>
#include <torch/device/Random.h>
#include <torch/device/Ray.h>
#include <torch/device/Sampling.h>

rtDeclareVariable(rtObject, sceneRoot, , );
rtDeclareVariable(float, sceneEpsilon, , );

rtDeclareVariable(float3, albedo, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(torch::RadianceData, rayData, rtPayload, );
rtDeclareVariable(float, hitDist, rtIntersectionDistance, );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );

rtDeclareVariable(optix::Ray, sray, rtCurrentRay, );
rtDeclareVariable(torch::ShadowData, srayData, rtPayload, );

typedef rtCallableProgramX<void(torch::LightSample&)> SampleLightFunction;
rtDeclareVariable(SampleLightFunction, SampleLights, , );

 TORCH_DEVICE optix::Matrix3x3 NormalToRotation(const float3& n)
 {
   float3 u = make_float3(1, 0, 0);
   float3 v = make_float3(0, 0, 1);
   if (dot(u, n) < dot(v, n)) u = v;
   v = normalize(cross(n, u));
   u = normalize(cross(n, v));

   optix::Matrix3x3 R;
   R.setCol(0, u);
   R.setCol(1, n);
   R.setCol(2, v);
   return R;
 }

TORCH_DEVICE void SampleBrdf(torch::BrdfSample& sample)
{
  float3 direction;
  torch::SampleHemisphereCosine(torch::randf2(sample.seed), direction);
  sample.pdf = direction.z / M_PIf;
  sample.direction = NormalToRotation(sample.normal) * direction;
  sample.throughput = 0.5 * albedo / M_PIf;
}

RT_PROGRAM void ClosestHit()
{
  torch::LightSample sample;
  sample.origin = ray.origin + hitDist * ray.direction;
  sample.tmin = sceneEpsilon;
  sample.seed = rayData.seed;

  SampleLights(sample);
  rayData.seed = sample.seed;

  float theta = dot(geometricNormal, sample.direction);

  if (theta > 0.0f)
  {
    optix::Ray shadowRay;
    shadowRay.origin = sample.origin;
    shadowRay.direction = sample.direction;
    shadowRay.ray_type = torch::RAY_TYPE_SHADOW;
    shadowRay.tmin = sample.tmin;
    shadowRay.tmax = sample.tmax;

    torch::ShadowData shadowData;
    shadowData.occluded = false;

    rtTrace(sceneRoot, shadowRay, shadowData);

    // if (!shadowData.occluded && rayData.depth > 0)
    // {
      float3 brdf = albedo / M_PIf;
      theta = dot(shadingNormal, sample.direction);
      rayData.radiance += rayData.throughput * brdf * sample.radiance * theta / sample.pdf;
    // }
  }

  torch::BrdfSample brdfSample;
  brdfSample.normal = shadingNormal;
  brdfSample.seed = rayData.seed;
  SampleBrdf(brdfSample);

  theta = dot(shadingNormal, brdfSample.direction);
  rayData.bounce.origin = sample.origin;
  rayData.bounce.direction = brdfSample.direction;
  rayData.bounce.throughput = make_float3(0.5); // theta * rayData.throughput * brdfSample.throughput / brdfSample.pdf;
}

RT_PROGRAM void AnyHit()
{
  srayData.occluded = true;
  rtTerminateRay();
}