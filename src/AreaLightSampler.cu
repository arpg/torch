#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include <torch/device/Light.h>
#include <torch/device/Geometry.h>
#include <torch/device/Random.h>

typedef rtCallableProgramX<unsigned int(float, float&)> SampleLightFunction;
typedef rtCallableProgramX<void(torch::GeometrySample&)> SampleGeomFunction;
rtDeclareVariable(SampleLightFunction, GetLightIndex, , );
rtDeclareVariable(SampleGeomFunction, SampleGeometry, , );
rtBuffer<torch::AreaLightData, 1> lights;

RT_CALLABLE_PROGRAM void Sample(torch::LightSample& sample)
{
  const float rand = torch::randf(sample.seed);
  const unsigned int index = GetLightIndex(rand, sample.pdf);
  const torch::AreaLightData& light = lights[index];

  torch::GeometrySample geomSample;
  geomSample.id = light.geometry;
  geomSample.origin = sample.origin;
  geomSample.tmin = sample.tmin;
  SampleGeometry(geomSample);

  const float3 difference = geomSample.position - sample.origin;
  sample.radiance = light.radiance;
  sample.direction = normalize(difference);
  sample.tmax = length(difference);
  sample.pdf *= geomSample.pdf;
}