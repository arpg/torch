#include "hip/hip_runtime.h"
#include <torch/device/Light.h>
#include <torch/device/Random.h>
#include <torch/device/Ray.h>

typedef rtCallableProgramX<void(torch::LightSample&)> SampleLightFunction;
rtDeclareVariable(SampleLightFunction, SampleLights, , );
rtDeclareVariable(float, sceneEpsilon, , );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(torch::RadianceData, rayData, rtPayload, );
rtDeclareVariable(unsigned int, launchIndex, rtLaunchIndex, );
rtDeclareVariable(unsigned int, sampleCount, , );
rtDeclareVariable(rtObject, dummyRoot, , );
rtBuffer<float3> vertices;
rtBuffer<float3> normals;
rtBuffer<float3> albedos;

TORCH_DEVICE void UpdateAlbedo(const float3& shading)
{
  // TODO: handle divide-by-zero better

  float3 albedo = albedos[launchIndex];

  if (shading.x > 1E-4) albedo.x /= shading.x; else albedo.x = 1.0f;
  if (shading.y > 1E-4) albedo.y /= shading.y; else albedo.y = 1.0f;
  if (shading.z > 1E-4) albedo.z /= shading.z; else albedo.z = 1.0f;

  albedos[launchIndex] = albedo;
}

TORCH_DEVICE float3 GetNormal()
{
  return normalize(normals[launchIndex]);
}

TORCH_DEVICE void InitializeRay(optix::Ray& ray)
{
  ray.origin = make_float3(0, 0, 0);
  ray.direction = make_float3(0, 0, 1);
  ray.ray_type = torch::RAY_TYPE_RADIANCE;
  ray.tmax = RT_DEFAULT_MAX;
  ray.tmin = 0.0f;
}

TORCH_DEVICE unsigned int InitializeSeed()
{
  return torch::init_seed<16>(launchIndex, 7919);
}

RT_PROGRAM void Remove()
{
  optix::Ray ray;
  InitializeRay(ray);
  torch::RadianceData data;
  data.seed = InitializeSeed();
  rtTrace(dummyRoot, ray, data);
}

RT_PROGRAM void ClosestHit()
{
  float3 shading = make_float3(0, 0, 0);
  const float throughput = 1.0f / (sampleCount * M_PIf);

  torch::LightSample sample;
  sample.origin = vertices[launchIndex];
  sample.tmin = sceneEpsilon;
  sample.seed = rayData.seed;
  sample.normal = GetNormal();
  sample.snormal = sample.normal;
  sample.throughput = make_float3(throughput);

  for (unsigned int i = 0; i < sampleCount; ++i)
  {
    SampleLights(sample);
    shading += sample.radiance;
  }

  UpdateAlbedo(shading);
}

RT_PROGRAM void Intersect(unsigned int index)
{
  if (rtPotentialIntersection(1.0f)) rtReportIntersection(0);
}

RT_PROGRAM void GetBounds(unsigned int index, float bounds[6])
{
  bounds[0] = -1; bounds[1] = -1; bounds[2] = -1;
  bounds[3] = +1; bounds[4] = +1; bounds[5] = +1;
}